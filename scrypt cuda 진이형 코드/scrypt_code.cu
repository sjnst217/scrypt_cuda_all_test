#include "hip/hip_runtime.h"
#include "type_origin.cuh"

typedef struct {
	uint32_t digest[8];
	uint64_t ptLen;
	uint8_t BUF[GPU_SHA256_BLOCK];
	uint32_t lastLen;
}SHA256_INFO;

typedef struct {
	uint32_t IPAD[8];
	uint32_t OPAD[8];
	uint64_t ptLen;
}PBKDF2_HMAC_SHA256_INFO;

#define GPU_ENDIAN_CHANGE32(X)		((GPU_rotl32((X),  8) & 0x00ff00ff) | (GPU_rotl32((X), 24) & 0xff00ff00))

__device__ void _SHA256_init(SHA256_INFO* info) {
	info->digest[0] = 0x6a09e667;
	info->digest[1] = 0xbb67ae85;
	info->digest[2] = 0x3c6ef372;
	info->digest[3] = 0xa54ff53a;
	info->digest[4] = 0x510e527f;
	info->digest[5] = 0x9b05688c;
	info->digest[6] = 0x1f83d9ab;
	info->digest[7] = 0x5be0cd19;

	for (int i = 0; i < GPU_SHA256_BLOCK; i++) {
		info->BUF[i] = 0;
	}
	info->ptLen = 0, info->lastLen = 0;
}
__device__ void _SHA256_core(uint32_t* input, uint32_t* digest) {
	uint32_t a, b, c, d, e, f, g, h;
	uint32_t w0_t = GPU_ENDIAN_CHANGE32(input[0]);
	uint32_t w1_t = GPU_ENDIAN_CHANGE32(input[1]);
	uint32_t w2_t = GPU_ENDIAN_CHANGE32(input[2]);
	uint32_t w3_t = GPU_ENDIAN_CHANGE32(input[3]);
	uint32_t w4_t = GPU_ENDIAN_CHANGE32(input[4]);
	uint32_t w5_t = GPU_ENDIAN_CHANGE32(input[5]);
	uint32_t w6_t = GPU_ENDIAN_CHANGE32(input[6]);
	uint32_t w7_t = GPU_ENDIAN_CHANGE32(input[7]);
	uint32_t w8_t = GPU_ENDIAN_CHANGE32(input[8]);
	uint32_t w9_t = GPU_ENDIAN_CHANGE32(input[9]);
	uint32_t wa_t = GPU_ENDIAN_CHANGE32(input[10]);
	uint32_t wb_t = GPU_ENDIAN_CHANGE32(input[11]);
	uint32_t wc_t = GPU_ENDIAN_CHANGE32(input[12]);
	uint32_t wd_t = GPU_ENDIAN_CHANGE32(input[13]);
	uint32_t we_t = GPU_ENDIAN_CHANGE32(input[14]);
	uint32_t wf_t = GPU_ENDIAN_CHANGE32(input[15]);

	a = digest[0];
	b = digest[1];
	c = digest[2];
	d = digest[3];
	e = digest[4];
	f = digest[5];
	g = digest[6];
	h = digest[7];

	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x428a2f98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x71374491);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0xb5c0fbcf);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0xe9b5dba5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x3956c25b);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x59f111f1);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x923f82a4);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0xab1c5ed5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xd807aa98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x12835b01);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x243185be);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x550c7dc3);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x72be5d74);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0x80deb1fe);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x9bdc06a7);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc19bf174);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0xe49b69c1);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0xefbe4786);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x0fc19dc6);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x240ca1cc);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x2de92c6f);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4a7484aa);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5cb0a9dc);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x76f988da);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x983e5152);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa831c66d);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xb00327c8);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xbf597fc7);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xc6e00bf3);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd5a79147);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x06ca6351);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x14292967);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x27b70a85);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x2e1b2138);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x4d2c6dfc);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x53380d13);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x650a7354);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x766a0abb);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x81c2c92e);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x92722c85);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xa2bfe8a1);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa81a664b);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xc24b8b70);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xc76c51a3);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xd192e819);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd6990624);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xf40e3585);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x106aa070);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x19a4c116);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x1e376c08);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x2748774c);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x34b0bcb5);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x391c0cb3);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4ed8aa4a);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5b9cca4f);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x682e6ff3);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x748f82ee);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x78a5636f);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x84c87814);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x8cc70208);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x90befffa);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xa4506ceb);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xbef9a3f7);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc67178f2);

	digest[0] += a;
	digest[1] += b;
	digest[2] += c;
	digest[3] += d;
	digest[4] += e;
	digest[5] += f;
	digest[6] += g;
	digest[7] += h;
}
__device__ void _SHA256_process(uint8_t* pt, uint64_t ptLen, SHA256_INFO* info) {
	uint64_t pt_index = 0;
	while ((ptLen + info->lastLen) >= GPU_SHA256_BLOCK) {
		for (int i = info->lastLen; i < (GPU_SHA256_BLOCK - info->lastLen); i++) {
			info->BUF[i] = pt[i + pt_index];
		}
		_SHA256_core((uint32_t*)info->BUF, info->digest);
		ptLen -= (GPU_SHA256_BLOCK - info->lastLen);
		info->ptLen += (GPU_SHA256_BLOCK - info->lastLen);
		pt_index += (GPU_SHA256_BLOCK - info->lastLen);
		info->lastLen = 0;
	}
	for (int i = 0; i < ptLen; i++)
		info->BUF[i + info->lastLen] = pt[i + pt_index];
	info->lastLen += ptLen;
}
__device__ void _SHA256_final(SHA256_INFO* info, uint8_t* out) {
	uint64_t r = (info->lastLen) % GPU_SHA256_BLOCK;
	info->BUF[r++] = 0x80;
	if (r >= GPU_SHA256_BLOCK - 8) {
		for (uint64_t i = r; i < GPU_SHA256_BLOCK; i++)
			info->BUF[i] = 0;
		_SHA256_core((uint32_t*)info->BUF, info->digest);
		for (int i = 0; i < GPU_SHA256_BLOCK - 8; i++)
			info->BUF[i] = 0;
	}
	else {
		for (uint64_t i = r; i < GPU_SHA256_BLOCK - 8; i++)
			info->BUF[i] = 0;
	}
	((uint32_t*)info->BUF)[GPU_SHA256_BLOCK / 4 - 2] = GPU_ENDIAN_CHANGE32((info->ptLen + info->lastLen) >> 29);
	((uint32_t*)info->BUF)[GPU_SHA256_BLOCK / 4 - 1] = GPU_ENDIAN_CHANGE32((info->ptLen + info->lastLen) << 3) & 0xffffffff;
	_SHA256_core((uint32_t*)info->BUF, info->digest);
	out[0] = (info->digest[0] >> 24) & 0xff;
	out[1] = (info->digest[0] >> 16) & 0xff;
	out[2] = (info->digest[0] >> 8) & 0xff;
	out[3] = (info->digest[0]) & 0xff;

	out[4] = (info->digest[1] >> 24) & 0xff;
	out[5] = (info->digest[1] >> 16) & 0xff;
	out[6] = (info->digest[1] >> 8) & 0xff;
	out[7] = (info->digest[1]) & 0xff;

	out[8] = (info->digest[2] >> 24) & 0xff;
	out[9] = (info->digest[2] >> 16) & 0xff;
	out[10] = (info->digest[2] >> 8) & 0xff;
	out[11] = (info->digest[2]) & 0xff;

	out[12] = (info->digest[3] >> 24) & 0xff;
	out[13] = (info->digest[3] >> 16) & 0xff;
	out[14] = (info->digest[3] >> 8) & 0xff;
	out[15] = (info->digest[3]) & 0xff;

	out[16] = (info->digest[4] >> 24) & 0xff;
	out[17] = (info->digest[4] >> 16) & 0xff;
	out[18] = (info->digest[4] >> 8) & 0xff;
	out[19] = (info->digest[4]) & 0xff;

	out[20] = (info->digest[5] >> 24) & 0xff;
	out[21] = (info->digest[5] >> 16) & 0xff;
	out[22] = (info->digest[5] >> 8) & 0xff;
	out[23] = (info->digest[5]) & 0xff;

	out[24] = (info->digest[6] >> 24) & 0xff;
	out[25] = (info->digest[6] >> 16) & 0xff;
	out[26] = (info->digest[6] >> 8) & 0xff;
	out[27] = (info->digest[6]) & 0xff;

	out[28] = (info->digest[7] >> 24) & 0xff;
	out[29] = (info->digest[7] >> 16) & 0xff;
	out[30] = (info->digest[7] >> 8) & 0xff;
	out[31] = (info->digest[7]) & 0xff;
}
__device__ void _SHA256(uint8_t* pt, uint64_t ptLen, uint8_t* digest) {
	SHA256_INFO info;
	_SHA256_init(&info);
	_SHA256_process(pt, ptLen, &info);
	_SHA256_final(&info, digest);
}
__device__ void _SHA256_preCompute_core(uint32_t* input, uint32_t* digest) {
	for (int i = 0; i < 16; i++)
		input[i] = GPU_ENDIAN_CHANGE32(input[i]);

	uint32_t w0_t = input[0];
	uint32_t w1_t = input[1];
	uint32_t w2_t = input[2];
	uint32_t w3_t = input[3];
	uint32_t w4_t = input[4];
	uint32_t w5_t = input[5];
	uint32_t w6_t = input[6];
	uint32_t w7_t = input[7];
	uint32_t w8_t = input[8];
	uint32_t w9_t = input[9];
	uint32_t wa_t = input[10];
	uint32_t wb_t = input[11];
	uint32_t wc_t = input[12];
	uint32_t wd_t = input[13];
	uint32_t we_t = input[14];
	uint32_t wf_t = input[15];


	uint32_t a = 0x6a09e667;
	uint32_t b = 0xbb67ae85;
	uint32_t c = 0x3c6ef372;
	uint32_t d = 0xa54ff53a;
	uint32_t e = 0x510e527f;
	uint32_t f = 0x9b05688c;
	uint32_t g = 0x1f83d9ab;
	uint32_t h = 0x5be0cd19;


	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x428a2f98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x71374491);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0xb5c0fbcf);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0xe9b5dba5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x3956c25b);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x59f111f1);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x923f82a4);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0xab1c5ed5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xd807aa98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x12835b01);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x243185be);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x550c7dc3);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x72be5d74);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0x80deb1fe);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x9bdc06a7);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc19bf174);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0xe49b69c1);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0xefbe4786);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x0fc19dc6);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x240ca1cc);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x2de92c6f);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4a7484aa);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5cb0a9dc);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x76f988da);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x983e5152);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa831c66d);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xb00327c8);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xbf597fc7);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xc6e00bf3);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd5a79147);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x06ca6351);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x14292967);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x27b70a85);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x2e1b2138);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x4d2c6dfc);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x53380d13);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x650a7354);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x766a0abb);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x81c2c92e);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x92722c85);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xa2bfe8a1);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa81a664b);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xc24b8b70);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xc76c51a3);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xd192e819);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd6990624);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xf40e3585);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x106aa070);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x19a4c116);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x1e376c08);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x2748774c);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x34b0bcb5);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x391c0cb3);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4ed8aa4a);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5b9cca4f);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x682e6ff3);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x748f82ee);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x78a5636f);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x84c87814);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x8cc70208);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x90befffa);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xa4506ceb);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xbef9a3f7);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc67178f2);

	digest[0] = a + 0x6a09e667;
	digest[1] = b + 0xbb67ae85;
	digest[2] = c + 0x3c6ef372;
	digest[3] = d + 0xa54ff53a;
	digest[4] = e + 0x510e527f;
	digest[5] = f + 0x9b05688c;
	digest[6] = g + 0x1f83d9ab;
	digest[7] = h + 0x5be0cd19;
}
__device__ void _SHA256_salt_compute_final(SHA256_INFO* info, uint32_t* out) {
	uint64_t r = (info->lastLen) % GPU_SHA256_BLOCK;
	info->BUF[r++] = 0x80;
	if (r >= GPU_SHA256_BLOCK - 8) {
		for (uint64_t i = r; i < GPU_SHA256_BLOCK; i++)
			info->BUF[i] = 0;
		_SHA256_core((uint32_t*)info->BUF, info->digest);
		for (int i = 0; i < GPU_SHA256_BLOCK - 8; i++)
			info->BUF[i] = 0;
	}
	else {
		for (uint64_t i = r; i < GPU_SHA256_BLOCK - 8; i++)
			info->BUF[i] = 0;
	}
	((uint32_t*)info->BUF)[GPU_SHA256_BLOCK / 4 - 2] = GPU_ENDIAN_CHANGE32((info->ptLen + info->lastLen) >> 29);
	((uint32_t*)info->BUF)[GPU_SHA256_BLOCK / 4 - 1] = GPU_ENDIAN_CHANGE32((info->ptLen + info->lastLen) << 3) & 0xffffffff;
	_SHA256_core((uint32_t*)info->BUF, info->digest);

	out[0] = info->digest[0];
	out[1] = info->digest[1];
	out[2] = info->digest[2];
	out[3] = info->digest[3];
	out[4] = info->digest[4];
	out[5] = info->digest[5];
	out[6] = info->digest[6];
	out[7] = info->digest[7];
}
__device__ void _PBKDF2_HMAC_SHA256_precompute(uint8_t* pt, uint8_t ptLen, PBKDF2_HMAC_SHA256_INFO* info) {
	uint8_t K1[GPU_SHA256_BLOCK];
	uint8_t K2[GPU_SHA256_BLOCK];

	for (int i = 0; i < ptLen; i++) {
		K1[i] = 0x36 ^ pt[i];
		K2[i] = 0x5c ^ pt[i];
	}
	for (int i = ptLen; i < GPU_SHA256_BLOCK; i++) {
		K1[i] = 0x36;
		K2[i] = 0x5c;
	}
	_SHA256_preCompute_core((uint32_t*)K1, info->IPAD);
	_SHA256_preCompute_core((uint32_t*)K2, info->OPAD);
}
__device__ void _PBKDF2_HMAC_SHA256_salt_compute(uint8_t* salt, uint64_t saLen, uint32_t integer, PBKDF2_HMAC_SHA256_INFO* INFO, uint32_t* out) {
	SHA256_INFO info;
	uint8_t temp[4] = { (integer >> 24) & 0xff, (integer >> 16) & 0xff, (integer >> 8) & 0xff, (integer & 0xff) };
	info.digest[0] = INFO->IPAD[0];
	info.digest[1] = INFO->IPAD[1];
	info.digest[2] = INFO->IPAD[2];
	info.digest[3] = INFO->IPAD[3];
	info.digest[4] = INFO->IPAD[4];
	info.digest[5] = INFO->IPAD[5];
	info.digest[6] = INFO->IPAD[6];
	info.digest[7] = INFO->IPAD[7];
	info.ptLen = 64;
	info.lastLen = 0;
	_SHA256_process(salt, saLen, &info);
	_SHA256_process(temp, 4, &info);
	_SHA256_salt_compute_final(&info, out);
}
__device__ void _PBKDF2_HMAC_SHA256_core(uint32_t* _prestate, uint32_t* digest, uint32_t* in) {

	uint32_t w0_t = in[0];
	uint32_t w1_t = in[1];
	uint32_t w2_t = in[2];
	uint32_t w3_t = in[3];
	uint32_t w4_t = in[4];
	uint32_t w5_t = in[5];
	uint32_t w6_t = in[6];
	uint32_t w7_t = in[7];
	uint32_t w8_t = 0x80000000;
	uint32_t w9_t = 0;
	uint32_t wa_t = 0;
	uint32_t wb_t = 0;
	uint32_t wc_t = 0;
	uint32_t wd_t = 0;
	uint32_t we_t = 0;
	uint32_t wf_t = (64 + 32) << 3;

	uint32_t a = _prestate[0];
	uint32_t b = _prestate[1];
	uint32_t c = _prestate[2];
	uint32_t d = _prestate[3];
	uint32_t e = _prestate[4];
	uint32_t f = _prestate[5];
	uint32_t g = _prestate[6];
	uint32_t h = _prestate[7];

	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x428a2f98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x71374491);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0xb5c0fbcf);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0xe9b5dba5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x3956c25b);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x59f111f1);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x923f82a4);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0xab1c5ed5);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xd807aa98);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x12835b01);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x243185be);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x550c7dc3);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x72be5d74);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0x80deb1fe);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x9bdc06a7);
	GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc19bf174);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0xe49b69c1);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0xefbe4786);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x0fc19dc6);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x240ca1cc);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x2de92c6f);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4a7484aa);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5cb0a9dc);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x76f988da);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x983e5152);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa831c66d);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xb00327c8);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xbf597fc7);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xc6e00bf3);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd5a79147);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0x06ca6351);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x14292967);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x27b70a85);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x2e1b2138);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x4d2c6dfc);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x53380d13);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x650a7354);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x766a0abb);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x81c2c92e);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x92722c85);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0xa2bfe8a1);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0xa81a664b);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0xc24b8b70);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0xc76c51a3);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0xd192e819);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xd6990624);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xf40e3585);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0x106aa070);

	w0_t = GPU_SHA256_EXPAND(we_t, w9_t, w1_t, w0_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, 0x19a4c116);
	w1_t = GPU_SHA256_EXPAND(wf_t, wa_t, w2_t, w1_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, 0x1e376c08);
	w2_t = GPU_SHA256_EXPAND(w0_t, wb_t, w3_t, w2_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, 0x2748774c);
	w3_t = GPU_SHA256_EXPAND(w1_t, wc_t, w4_t, w3_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, 0x34b0bcb5);
	w4_t = GPU_SHA256_EXPAND(w2_t, wd_t, w5_t, w4_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, 0x391c0cb3);
	w5_t = GPU_SHA256_EXPAND(w3_t, we_t, w6_t, w5_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, 0x4ed8aa4a);
	w6_t = GPU_SHA256_EXPAND(w4_t, wf_t, w7_t, w6_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, 0x5b9cca4f);
	w7_t = GPU_SHA256_EXPAND(w5_t, w0_t, w8_t, w7_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, 0x682e6ff3);
	w8_t = GPU_SHA256_EXPAND(w6_t, w1_t, w9_t, w8_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, 0x748f82ee);
	w9_t = GPU_SHA256_EXPAND(w7_t, w2_t, wa_t, w9_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, 0x78a5636f);
	wa_t = GPU_SHA256_EXPAND(w8_t, w3_t, wb_t, wa_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, 0x84c87814);
	wb_t = GPU_SHA256_EXPAND(w9_t, w4_t, wc_t, wb_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, 0x8cc70208);
	wc_t = GPU_SHA256_EXPAND(wa_t, w5_t, wd_t, wc_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, 0x90befffa);
	wd_t = GPU_SHA256_EXPAND(wb_t, w6_t, we_t, wd_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, 0xa4506ceb);
	we_t = GPU_SHA256_EXPAND(wc_t, w7_t, wf_t, we_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, c, d, e, f, g, h, a, b, we_t, 0xbef9a3f7);
	wf_t = GPU_SHA256_EXPAND(wd_t, w8_t, w0_t, wf_t); GPU_SHA256_STEP(GPU_SHA256_F0o, GPU_SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, 0xc67178f2);

	digest[0] = _prestate[0] + a;
	digest[1] = _prestate[1] + b;
	digest[2] = _prestate[2] + c;
	digest[3] = _prestate[3] + d;
	digest[4] = _prestate[4] + e;
	digest[5] = _prestate[5] + f;
	digest[6] = _prestate[6] + g;
	digest[7] = _prestate[7] + h;
}
__device__ void PBKDF2_HMAC_SHA256(uint8_t* pt, size_t ptLen, uint8_t* salt, size_t saLen, uint8_t* dkout, size_t dkLen, size_t iter) {
	uint8_t buf[GPU_SHA256_BLOCK];
	uint32_t _first[8];
	uint32_t _second[8];
	PBKDF2_HMAC_SHA256_INFO info;
	uint32_t _TkLen = dkLen / GPU_SHA256_DIGEST;
	if (dkLen % GPU_SHA256_DIGEST != 0) { _TkLen++; }


	if (ptLen > GPU_SHA256_BLOCK) {
		_SHA256(pt, ptLen, buf);
		_PBKDF2_HMAC_SHA256_precompute(buf, GPU_SHA256_DIGEST, &info);
		info.ptLen = GPU_SHA256_DIGEST;
	}
	else {
		_PBKDF2_HMAC_SHA256_precompute(pt, ptLen, &info);
		info.ptLen = ptLen;
	}
	for (uint32_t i = 0; i < _TkLen; i++) {
		_PBKDF2_HMAC_SHA256_salt_compute(salt, saLen, i + 1, &info, _first);
		_PBKDF2_HMAC_SHA256_core(info.OPAD, _second, _first);
		for (int z = 0; z < 8; z++) {
			dkout[4 * z + 0] = (_second[z] >> 24) & 0xff;
			dkout[4 * z + 1] = (_second[z] >> 16) & 0xff;
			dkout[4 * z + 2] = (_second[z] >> 8) & 0xff;
			dkout[4 * z + 3] = (_second[z] >> 0) & 0xff;
		}
		dkout += 32;
	}
}


__device__ void salsa208_word_specification(uint32_t inout[16])
{
	int i;
	uint32_t x[16];
	memcpy(x, inout, sizeof(uint32_t) * 16);
	for (i = 8; i > 0; i -= 2) {
		x[4] ^= R(x[0] + x[12], 7);
		x[8] ^= R(x[4] + x[0], 9);
		x[12] ^= R(x[8] + x[4], 13);
		x[0] ^= R(x[12] + x[8], 18);
		x[9] ^= R(x[5] + x[1], 7);
		x[13] ^= R(x[9] + x[5], 9);
		x[1] ^= R(x[13] + x[9], 13);
		x[5] ^= R(x[1] + x[13], 18);
		x[14] ^= R(x[10] + x[6], 7);
		x[2] ^= R(x[14] + x[10], 9);
		x[6] ^= R(x[2] + x[14], 13);
		x[10] ^= R(x[6] + x[2], 18);
		x[3] ^= R(x[15] + x[11], 7);
		x[7] ^= R(x[3] + x[15], 9);
		x[11] ^= R(x[7] + x[3], 13);
		x[15] ^= R(x[11] + x[7], 18);
		x[1] ^= R(x[0] + x[3], 7);
		x[2] ^= R(x[1] + x[0], 9);
		x[3] ^= R(x[2] + x[1], 13);
		x[0] ^= R(x[3] + x[2], 18);
		x[6] ^= R(x[5] + x[4], 7);
		x[7] ^= R(x[6] + x[5], 9);
		x[4] ^= R(x[7] + x[6], 13);
		x[5] ^= R(x[4] + x[7], 18);
		x[11] ^= R(x[10] + x[9], 7);
		x[8] ^= R(x[11] + x[10], 9);
		x[9] ^= R(x[8] + x[11], 13);
		x[10] ^= R(x[9] + x[8], 18);
		x[12] ^= R(x[15] + x[14], 7);
		x[13] ^= R(x[12] + x[15], 9);
		x[14] ^= R(x[13] + x[12], 13);
		x[15] ^= R(x[14] + x[13], 18);
	}
	for (i = 0; i < 16; ++i)
		inout[i] += x[i];
}
__device__ void scryptBlockMix(uint32_t* B_, uint32_t* B, uint64_t r) // B_�� ���� B�� ���� BlockMix���� ���� ��������
{
	uint64_t i, j;
	uint32_t X[16], * pB;

	memcpy(X, B + (r * 2 - 1) * 16, sizeof(uint32_t) * 16);
	pB = B;
	for (i = 0; i < r * 2; i++) {
		for (j = 0; j < 16; j++)
			X[j] ^= *pB++;
		salsa208_word_specification(X);
		memcpy(B_ + (i / 2 + (i & 1) * r) * 16, X, sizeof(X));
	}
} 
__device__ void scryptROMix(unsigned char* B, uint64_t r, uint64_t N, uint32_t* X, uint32_t* T, uint32_t* V)
{
	unsigned char* pB;
	uint32_t* pV;
	uint64_t i, k;
	uint64_t cycle0 = 0;
	uint64_t cycle1 = 0;
	/* Convert from little endian input */
	for (pV = V, i = 0, pB = B; i < 32 * r; i++, pV++) {	//unsigned int ���� pV���ٰ� unsigned char ���� pB�� ����־��ֱ�
		*pV = *pB++;										//-> �̷��� 128 * r �� �ƴ϶� 32 * r �� �ٲ� �� �ִ� -> �ݺ� Ƚ���� ���� �� ����
		*pV |= *pB++ << 8;									// �ᱹ �� ó�� V�� ����� �ִ� �ڵ�
		*pV |= *pB++ << 16;
		*pV |= (uint32_t)*pB++ << 24;
	}
	for (i = 1; i < N; i++, pV += 32 * r)					//V�� N - 1 ���� �������ִ� �ڵ�
		scryptBlockMix(pV, pV - 32 * r, r);
	scryptBlockMix(X, V + (N - 1) * 32 * r, r);

	for (i = 0; i < N; i++) {
		uint32_t j;						//mod������ ���ؼ� ���° V���� ����� �� �������ֱ� ���� ����
		j = X[16 * (2 * r - 1)] % N;	//mod������ ���ְ�
		pV = V + 32 * r * j;			//pV�� ���� V�� j��° �� �־��ְ�
		for (k = 0; k < 32 * r; k++)
			T[k] = X[k] ^ *pV++;		//T�� ���� X�� pV�� �� XOR����
		scryptBlockMix(X, T, r);		//X�� ���� T�� ���� blockMix�ؼ� ����
	}


	/* Convert output to little endian */
	for (i = 0, pB = B; i < 32 * r; i++) { //pB�� ������ ���� ������� ����־��ֱ� ���ؼ� endian ��ȯ�� ���ؼ� ����
		uint32_t xtmp = X[i];
		*pB++ = xtmp & 0xff;
		*pB++ = (xtmp >> 8) & 0xff;
		*pB++ = (xtmp >> 16) & 0xff;
		*pB++ = (xtmp >> 24) & 0xff;
	}
}


//				������� ��ü �Ҵ� block, ��ü �Ҵ� password, password�� ����, ��ü �Ҵ� salt, salt�� ����, N, r, p, ��ü �Ҵ� key, key�� ����
__global__ void GPU_scrypt(uint8_t* B, uint8_t* pass, size_t passlen, uint8_t* salt, size_t saltlen, uint64_t N, uint64_t r, uint64_t p, uint8_t* key, size_t keylen) 
{
	uint64_t data_index = 0;											//���� �������� block(data)�� ���� index
	uint64_t store_index = 0;										//���� �������� key�� ������ �� ����ϱ� ���� index
	uint64_t i, Blen, Vlen, j;
	uint64_t total_len;
	Blen = p * 128 * r;													//block ��ü�� ����
	Vlen = 32 * r * (N + 2) * sizeof(uint32_t);					//���ο��� ����ϴ� Vector�� ����
	total_len = Blen + Vlen;											//���Ǵ� ��� ������ ��
	data_index = (blockDim.x * blockIdx.x) + threadIdx.x;	//������ ������ ��ȣ�� ���ؼ�
	data_index = data_index * total_len;							//��ü data ���̿� �����ִ� ������ ���� �������� block�� ���� index�� ������
	store_index = (blockDim.x * blockIdx.x) + threadIdx.x;	//������ ������ ��ȣ�� ���ؼ�
	store_index = store_index * keylen;							//��ü key ���̿� �����ִ� ������ ���� �������� key�� ���� index�� ������

	uint32_t* X = NULL;												//���� �����忡�� ���� X(ROMix�Լ��� �ʱ� block)�� ���� �����ϱ� ���� ����
	uint32_t* T = NULL;												//���� �����忡�� ���� T(ROMix�Լ��� �߰� ��� block (X XOR V_j ���� �ֵ�))�� ���� �����ϱ� ���� ����
	uint32_t* V = NULL;												//���� �����忡�� ���� V(Vector)�� ���� �����ϱ� ���� ����
	X = (uint32_t*)(B + Blen + data_index);
	T = X + 32 * r;
	V = T + 32 * r;

	PBKDF2_HMAC_SHA256(pass + (passlen * (blockDim.x * blockIdx.x + threadIdx.x)), passlen, salt + (saltlen * (blockDim.x * blockIdx.x + threadIdx.x)), saltlen, B + data_index, Blen, 1);
	for (i = 0; i < p; i++)	
		scryptROMix(B + data_index + 128 * r * i, r, N, X, T, V);
	PBKDF2_HMAC_SHA256(pass + (passlen * (blockDim.x * blockIdx.x + threadIdx.x)), passlen, B + data_index, Blen, key + store_index, keylen, 1);
}
void performance_test_scrypt(uint32_t blocksize, uint32_t threadsize) {
	hipError_t err;
	hipEvent_t start, stop;
	uint8_t password[8] = { 0x70, 0x61, 0x73, 0x73, 0x77, 0x6f, 0x72, 0x64 };
	uint8_t salt[4] = { 0x4e, 0x61, 0x43, 0x6c };
	uint8_t* cpu_key = (uint8_t*)malloc(blocksize * threadsize * 64); //scrypt ���� ���� key����� 
	if (cpu_key == NULL)
		return;

	uint8_t* gpu_pass = NULL;										//gpu ���ο����� password
	uint8_t* gpu_salt = NULL;										//gpu ���ο����� salt
	uint8_t* gpu_key = NULL;										//gpu ���ο����� key
	uint8_t* gpu_b = NULL;											//gpu ���ο����� ��ü block
	uint64_t Blen = 128 * 2 * 8;									//block ���� (128 * p * r)
	uint64_t Vlen = 32 * 8 * (1024 + 2) * sizeof(uint32_t);			//���ο��� ����ϴ� Vector�� ���� (128 * r * N) ���⿡�� 32�� ������ Vector�� ������ �� uint8_t ���� block�� uint32_t�� �ٲپ �����ϱ� ����, (N + 2)�� �� �� ������ X block �� ������ �����(output) �� ũ�⸦ �Ҵ��ϱ� ���ؼ� �ε�?
	uint64_t total = Blen + Vlen;									//���ο� �Ҵ��� ��ü ũ��
	float elapsed_time_ms = 0.0f;


	hipMalloc((void**)&gpu_pass, 8 * blocksize * threadsize);			//gpu ��ü password �Ҵ�
	hipMalloc((void**)&gpu_salt, 4 * blocksize * threadsize);			//gpu ��ü salt �Ҵ�
	hipMalloc((void**)&gpu_key, 64 * blocksize * threadsize);			//gpu ��ü key �Ҵ�
	err = hipMalloc((void**)&gpu_b, total * blocksize * threadsize);	//gpu ��ü block �Ҵ� -> ��ü block ũ�⸦ �Ҵ��ؾ� �ϱ� ������ ���Ǵ� ��� block�� ũ�⸦ �����ؼ� �Ҵ��� �־�� �� 
	if (err != hipSuccess) {
		printf("gpu_b : CUDA error : %s\n", hipGetErrorString(err));
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	for (int i = 0; i < blocksize * threadsize; i++) {
		hipMemcpy(gpu_pass + (8 * i), password, 8, hipMemcpyHostToDevice);	//gpu_pass���� ������ �����忡�� ����� ������ password�� �������� -> ��ȯ�ϰ��� �ϴ� �ڵ忡���� 1���� ���ָ� ��
		hipMemcpy(gpu_salt + (4 * i), salt, 4, hipMemcpyHostToDevice);		//gpu_salt���� ������ �����忡�� ����� ������ salt�� �������� -> ���� ����
		password[7] = (i + 1) &0xff;											//�� �� �ݺ� ������ password�� ��ȯ�����ֱ� ���� -> ��ȯ�ϰ��� �ϴ� �ڵ忡���� �����൵ ��
		salt[3] = (i + 2) &0xff;												//�� �� �ݺ� ������ salt�� ��ȯ�����ֱ� ���� -> ���� ����
	}

	for (int i = 0; i < 1; i++) {
		GPU_scrypt << <blocksize, threadsize >> > (gpu_b, gpu_pass, 8, gpu_salt, 4, 1024, 8, 2, gpu_key, 64);	// ������� ��ü �Ҵ� block, ��ü �Ҵ� password, password�� ����, ��ü �Ҵ� salt, salt�� ����, N, r, p, ��ü �Ҵ� key, key�� ����
		hipMemcpy(cpu_key, gpu_key, 64 * blocksize * threadsize, hipMemcpyDeviceToHost);						// ������ ��ģ gpu�� ��ü key�� cpu�� ����
	}

	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	printf("%4.2f\n", elapsed_time_ms);
	printf("blocksize: %d, threadsize: %d, scrypt/s: %4.2f\n", blocksize, threadsize, blocksize * threadsize * (1000 / elapsed_time_ms));

	hipFree(gpu_pass);
	hipFree(gpu_salt);
	hipFree(gpu_b);
	hipFree(gpu_key);
	free(cpu_key);
}

int main() {
	performance_test_scrypt(2, 16);
	performance_test_scrypt(16, 2);

	performance_test_scrypt(2, 32);
	performance_test_scrypt(32, 2);

	performance_test_scrypt(2, 64);
	performance_test_scrypt(64, 2);

	performance_test_scrypt(2, 128);
	performance_test_scrypt(128, 2);

	performance_test_scrypt(2, 256);
	performance_test_scrypt(256, 2);



}